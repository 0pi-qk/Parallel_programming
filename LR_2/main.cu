#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <omp.h>

using namespace std;

void SaveFileData(int* data, int size_array) {
	ofstream fout("Result.txt", ios::app);
	fout << "___";
	for (int i = 0; i < size_array; i++)
		fout << data[i] << " ";
	fout << endl;
	fout.close();
}

void SaveFileSeparator() {
	ofstream fout("Result.txt", ios::app);
	fout << endl << "====== 1 - Initial; 2 - Parallel; 3 - Single ======" << endl;
	fout.close();
}

__device__ void ParallelSwap(int* a, int* b) {
	const int t = *a;
	*a = *b;
	*b = t;
}

__device__ void ParallelHeapify(int* maxHeap, int heapSize, int idx) {
	int largest = idx;
	int left = 2 * idx + 1;
	int right = 2 * idx + 2;
	if (left < heapSize && maxHeap[left] > maxHeap[largest]) {
		largest = left;
	}

	if (right < heapSize && maxHeap[right] > maxHeap[largest]) {
		largest = right;
	}

	if (largest != idx) {
		ParallelSwap(&maxHeap[idx], &maxHeap[largest]);
		ParallelHeapify(maxHeap, heapSize, largest);
	}
}

__global__ void ParallelSort(int* iA, const int size_array) {
	iA = iA + blockIdx.x * size_array;

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = size_array / 2 - 1; i >= 0; i--)
			ParallelHeapify(iA, size_array, i);

		for (int i = size_array - 1; i >= 0; i--) {
			ParallelSwap(&iA[0], &iA[i]);
			ParallelHeapify(iA, i, 0);
		}
	}
}

void SingleSwap(int* a, int* b) {
	const int t = *a;
	*a = *b;
	*b = t;
}

void SingleHeapify(int data[], int size_array, int i) {
	int largest = i;
	int l = 2 * i + 1;
	int r = 2 * i + 2;

	if (l < size_array && data[l] > data[largest])
		largest = l;

	if (r < size_array && data[r] > data[largest])
		largest = r;

	if (largest != i) {
		SingleSwap(&data[i], &data[largest]);
		SingleHeapify(data, size_array, largest);
	}
}

void SingleSort(int data[], int size_array) {
	for (int i = size_array / 2 - 1; i >= 0; i--)
		SingleHeapify(data, size_array, i);

	for (int i = size_array - 1; i >= 0; i--) {
		SingleSwap(&data[0], &data[i]);
		SingleHeapify(data, i, 0);
	}
}

bool ComparisonResults(int* single_mass, int* parallel_mass, int size_array) {
	for (int i = 0; i < size_array; i++)
		if (single_mass[i] != parallel_mass[i])
			return false;
	return true;
}

void StartTest(int size_array, bool save) {
	srand((unsigned)time(NULL));

	if (save == true) SaveFileSeparator();

	double parallel_time, single_time;

	int* data_parallel = (int*)malloc(size_array * sizeof(int));
	int* data_single = (int*)malloc(size_array * sizeof(int));

	int* temp_data_parallel = NULL;
	hipMalloc((void**)&temp_data_parallel, size_array * sizeof(int));

	for (int i = 0; i < size_array; ++i)
		data_single[i] = data_parallel[i] = rand() % 100 + 1;

	if (save == true) SaveFileData(data_parallel, size_array);

	hipMemcpy(temp_data_parallel, data_parallel, size_array * sizeof(int), hipMemcpyHostToDevice);

	int threads, blocks;

	if (size_array < 256) {
		threads = 1024;
		blocks = 1024;
	}
	else {
		blocks = 262144 / size_array;
		if (blocks < 32) threads = 32;
		else threads = 262144 / size_array;
	}

	parallel_time = omp_get_wtime();
	ParallelSort << <blocks, threads >> > (temp_data_parallel, size_array);
	parallel_time = omp_get_wtime() - parallel_time;
	hipDeviceSynchronize();

	hipMemcpy(data_parallel, temp_data_parallel, size_array * sizeof(int), hipMemcpyDeviceToHost);

	single_time = omp_get_wtime();
	SingleSort(data_single, size_array);
	single_time = omp_get_wtime() - single_time;

	if (ComparisonResults(data_parallel, data_single, size_array))
		cout << "Is equal" << endl;
	else
		cout << "Is not equal" << endl;

	printf("Time of single sorting:   %f\n", single_time);
	printf("Time of parallel sorting: %f\n", parallel_time);

	hipFree(temp_data_parallel);

	if (save == true) {
		SaveFileData(data_parallel, size_array);
		SaveFileData(data_single, size_array);
	}

	free(data_parallel);
	free(data_single);

	hipDeviceReset();
}

int main() {
    bool menu = true, save;
    int key, size_array;
    while (menu) {
        cout << "1|  Manual test mode " << endl;
        cout << "2|  Automatic test mode" << endl;
        cout << "3|  Out" << endl;
        cout << "-> ";
        cin >> key;
        cout << "==================================================";
        switch (key) {
        case 1: {
            cout << endl << "Enter list size - ";
            cin >> size_array;
            cout << "Save result to file? (0 or 1) - ";
            cin >> save;
            StartTest(size_array, save);
            cout << "==================================================" << endl;
            break;
        }
        case 2: {
            for (size_array = 10; size_array < 9000001;) {
                cout << endl << "List size - " << size_array << endl;
                StartTest(size_array, false);
                cout << "==================================================";
                if (size_array < 1000000) size_array *= 10;
                else size_array += 1000000;
            }
            cout << endl;
            break;
        }
        case 3: {
            menu = false;
            break;
        }
        default:
            break;
        }
    }
}